#include "hip/hip_runtime.h"
#include "framework.h"
#include <stdio.h>

//#define NAIVE

// todo: add declarations in cuh; reclamp 
__device__ int calculate_grid_index(glm::vec2 pos)
{
	int gridX = (int)glm::floor((pos.x - LEFT_WALL) / GRID_R);
	int gridY = (int)glm::floor((pos.y - DOWN_WALL) / GRID_R);

	//printf("(%f - %f) / %f = %f | %d\n", pos.x, LEFT_WALL, GRID_R, (pos.x - LEFT_WALL) / GRID_R, gridX);

	int gridSize = (int)glm::ceil(WORLD_WIDTH / GRID_R);
	gridX = glm::clamp(gridX, 0, gridSize - 1);
	gridY = glm::clamp(gridY, 0, gridSize - 1);

	return gridY * gridSize + gridX;
}

struct boidParamsStruct {
	glm::vec2* separation_component;
	glm::vec2* velocity_sum;
	glm::vec2* position_sum;
	int* neighbors;
} boidParams;

// todo: everything to const ref
__device__ void iterate_through_cell(const cudaArrays& soa, int cell, int i, boidParamsStruct boidParams)
{
	glm::vec2* pos = soa.positions;
	glm::vec2* pos_bb = soa.positions_bb;
	glm::vec2* vel = soa.velocities;
	int* grid_starts = soa.grid_starts;
	int* grid_cells = soa.grid_cells;
	int* grid_ends = soa.grid_ends;
	int* boids = soa.grid_boids;

	// todo: merge R with visib_radius
	float radius_sq = R * R;

	int start = grid_starts[cell];

	if (start == -1) // empty cell
		return;

	//int len = grid_cellsizes[cell];
	int end = grid_ends[cell];
	for (int k = start; k < end; k++)
	{
		int j = boids[k];
		if (i == j) continue;

		glm::vec2 diff = pos[i] - pos[j];
		//float lensq = glm::dot(diff, diff);
		float len = glm::length(diff);
		glm::vec2 norm = glm::normalize(diff);

		if (len < R)//)lensq < radius_sq)
		{
      //soa.velocities_bb[boids[k]] = glm::vec2(1, 0);
			(*boidParams.separation_component) += norm / len;
			(*boidParams.velocity_sum) += vel[j];
			(*boidParams.position_sum) += pos[j];
			(*boidParams.neighbors)++;
		}
	}
}

__device__ glm::vec2 apply_boid_rules(cudaArrays soa, const cpu_shoal::paramsStruct& params, int i, double d)
{
	glm::vec2* pos = soa.positions;
	glm::vec2* pos_bb = soa.positions_bb;
	glm::vec2* vel = soa.velocities;
	int* grid_starts = soa.grid_starts;
	int* grid_cells = soa.grid_cells;
	int* boids = soa.grid_boids;

	glm::vec2 separation_component(0, 0);
	glm::vec2 velocity_sum(0, 0);
	glm::vec2 position_sum(0, 0);
	int neighbors = 0;
	float radius_sq = R * R;

	boidParamsStruct boidParams;

	boidParams.separation_component = &separation_component;
	boidParams.velocity_sum = &velocity_sum;
	boidParams.position_sum = &position_sum;
	boidParams.neighbors = &neighbors;

#ifdef NAIVE
	for (int j = 0; j < N; ++j)
	{
		float len = glm::length(pos[i] - pos[j]);
		if (i != j && len < R)
		{
			separation_component += pos[i] - pos[j];
			velocity_sum += vel[j];
			position_sum += pos[j];

			neighbors++;
		}
	}
#else
	int density = (int)glm::ceil(WORLD_WIDTH / GRID_R);
	int cell = calculate_grid_index(pos[i]);
	int gridX = cell % density;
	int gridY = cell / density;

	// todo: calculate this beforehand? sort a duplicate buffer?
	int start_offset, horizontal_offset, vertical_offset;

	if (pos[i].x >= LEFT_WALL + (gridX + 0.5) * GRID_R)
	{
		start_offset = 0;
		horizontal_offset = 1;
	}
	else
	{
		start_offset = -1;
		horizontal_offset = -1;
	}

	if (pos[i].y >= DOWN_WALL + (gridY + 0.5) * GRID_R)
		vertical_offset = density;
	else
		vertical_offset = -density;

	iterate_through_cell(soa, cell, i, boidParams);

	if ((cell + horizontal_offset) / density == cell / density)
		iterate_through_cell(soa, cell + horizontal_offset, i, boidParams);

	if (cell + vertical_offset >= 0 && cell + vertical_offset < density * density)
		iterate_through_cell(soa, cell + vertical_offset, i, boidParams);

	if (cell + vertical_offset + horizontal_offset >= 0
		&& cell + vertical_offset + horizontal_offset < density * density
		&& (cell + vertical_offset + horizontal_offset) / density == (cell + vertical_offset) / density)
		iterate_through_cell(soa, cell + vertical_offset + horizontal_offset, i, boidParams);
#endif

	glm::vec2 alignment_component;
	glm::vec2 cohesion_component;

	if (neighbors == 0)
		return glm::vec2(0, 0);

	velocity_sum /= neighbors;
	position_sum /= neighbors;
	alignment_component = velocity_sum - vel[i];
	cohesion_component = position_sum - pos[i];

	return (float)d *
		(params.s * separation_component
			+ params.a * alignment_component
			+ params.c * cohesion_component);
}

__device__ glm::vec2 speed_limit(glm::vec2 vel, const cpu_shoal::paramsStruct& params)
{
	if (glm::length(vel) < params.min_speed)
		return params.min_speed * glm::normalize(vel);
	if (glm::length(vel) > params.max_speed)
		return params.max_speed * glm::normalize(vel);

	return vel;
}

__device__ glm::vec2 turn_from_wall(glm::vec2 pos, glm::vec2 vel, const cpu_shoal::paramsStruct& params)
{
	float dx_right = 1 - pos.x;
	float dx_left = pos.x + 1;
	float dy_up = 1 - pos.y;
	float dy_down = pos.y + 1;

	float len = glm::length(vel);

	glm::vec2 vel_change = glm::vec2(0, 0);

	if (dx_right < params.margin)
		vel_change.x -= params.turn * len / (dx_right * dx_right);
	if (dx_left < params.margin)
		vel_change.x += params.turn * len / (dx_left * dx_left);
	if (dy_up < params.margin)
		vel_change.y -= params.turn * len / (dy_up * dy_up);
	if (dy_down < params.margin)
		vel_change.y += params.turn * len / (dy_down * dy_down);

	return vel + vel_change;
}

__device__ glm::vec2 teleport_through_wall(glm::vec2 pos)
{
	glm::vec2 ret = pos;

	if (pos.x > 1)
		ret.x = -1;
	if (pos.y > 1)
		ret.y = -1;
	if (pos.x < -1)
		ret.x = 1;
	if (pos.y < -1)
		ret.y = 1;

	return ret;
}

__global__ void calculateGridKernel(cudaArrays soa)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i >= N)
		return;

	soa.grid_cells[i] = calculate_grid_index(soa.positions[i]);
	soa.grid_boids[i] = i;

	// these are getting sorted later
}

__global__ void calculateGridStartsKernel(struct cudaArrays soa)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i >= N)
		return;

	size_t density = (int)glm::ceil(WORLD_WIDTH / GRID_R);
	size_t grid_size = density * density * sizeof(int);

	int cell = soa.grid_cells[i];
	int prev_cell = i != 0 ? soa.grid_cells[i - 1] : -1;
	if (i == 0 || cell != prev_cell)
	{
		// todo: there's a bottleneck
		soa.grid_starts[cell] = i;
		if (i != 0)
			soa.grid_ends[prev_cell] = i;
	}
}

__global__ void calculateBoidsKernel(cudaArrays soa, cpu_shoal::paramsStruct params, double d, float x, float y)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i >= N)
		return;

#ifdef SNAP_TO_GRID
	glm::vec2* pos = soa.positions;
	glm::vec2* vel = soa.velocities;
	int* grid_starts = soa.grid_starts;
	int* grid_cells = soa.grid_cells;
	int* grid_cellsizes = soa.grid_cellsizes;
	int* boids = soa.grid_boids;

	int cell = calculate_grid_index(pos[i]);
	int density = (int)glm::ceil(WORLD_WIDTH / GRID_R);
	int gridX = cell % density;
	int gridY = cell / density;

	gridX = (int)glm::floor((pos[i].x - LEFT_WALL + 1e-3) / GRID_R);
	gridY = (int)glm::floor((pos[i].y - DOWN_WALL + 1e-3) / GRID_R);

	if (i != 42)
	{
		//soa.positions_bb[i] = glm::vec2(LEFT_WALL + GRID_R * gridX, DOWN_WALL + GRID_R * gridY);
	}
	else
		soa.positions_bb[i] = glm::vec2(x, y);//0.7f * glm::vec2(glm::sin(d), glm::cos(d));

	apply_boid_rules(soa, params, i, 1);
#else
	glm::vec2 new_vel;

	new_vel = soa.velocities[i]
		+ apply_boid_rules(soa, params, i, 1);
	new_vel = speed_limit(new_vel, params);
	new_vel = turn_from_wall(soa.positions[i], new_vel, params);

	soa.velocities_bb[i] = new_vel;
	glm::vec2 new_pos = soa.positions[i] + (float)d * new_vel;
	new_pos = teleport_through_wall(new_pos);

	soa.positions_bb[i] = new_pos;

	if (new_pos.x < -1 || new_pos.x > 1 || new_pos.y < -1 || new_pos.y > 1)
		printf("%f, %f\n", new_pos.x, new_pos.y);
#endif
	__syncthreads();

	soa.positions[i] = soa.positions_bb[i];
	soa.velocities[i] = soa.velocities_bb[i];

	glm::vec2 v = glm::normalize(soa.velocities[i]);
	glm::vec2 vT = glm::vec2(v.y, -v.x);
	soa.models[i] = glm::mat3(glm::vec3(v, 0), glm::vec3(vT, 0), glm::vec3(soa.positions[i], 1.0f));
}
