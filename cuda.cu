#include "hip/hip_runtime.h"
#include "framework.h"
#include <stdio.h>

//#define NAIVE
#define SNAP_TO_GRID

// todo: add declarations in cuh; reclamp 
__device__ int calculate_grid_index(glm::vec3 pos)
{
	int gridX = (int)glm::floor((pos.x - LEFT_WALL) / GRID_R);
	int gridY = (int)glm::floor((pos.y - DOWN_WALL) / GRID_R);
	int gridZ = (int)glm::floor((pos.z - BACK_WALL) / GRID_R);

	//printf("(%f - %f) / %f = %f | %d\n", pos.x, LEFT_WALL, GRID_R, (pos.x - LEFT_WALL) / GRID_R, gridX);

	int gridSize = (int)glm::ceil(WORLD_WIDTH / GRID_R);
	gridX = glm::clamp(gridX, 0, gridSize - 1);
	gridY = glm::clamp(gridY, 0, gridSize - 1);
	gridZ = glm::clamp(gridZ, 0, gridSize - 1);

	return gridZ * gridSize * gridSize
		+ gridY * gridSize
		+ gridX;
}

struct boidParamsStruct {
	glm::vec3* separation_component;
	glm::vec3* velocity_sum;
	glm::vec3* position_sum;
	int* neighbors;
} boidParams;

// todo: everything to const ref
__device__ void iterate_through_cell(const cudaArrays& soa, int cell, int i, boidParamsStruct boidParams)
{
	glm::vec3* pos = soa.positions;
	glm::vec3* pos_bb = soa.positions_bb;
	glm::vec3* vel = soa.velocities;
	int* grid_starts = soa.grid_starts;
	int* grid_cells = soa.grid_cells;
	int* grid_ends = soa.grid_ends;
	int* boids = soa.grid_boids;

	// todo: merge R with visib_radius
	float radius_sq = MIN_R * MIN_R;

	int start = grid_starts[cell];

	if (start == -1) // empty cell
		return;

	//int len = grid_cellsizes[cell];
	int end = grid_ends[cell];
	for (int k = start; k < end; k++)
	{
		int j = boids[k];
		if (i == j) continue;

		glm::vec3 diff = pos[i] - pos[j];
		float len = glm::length(diff);
		glm::vec3 norm = glm::normalize(diff);

		if (len < MIN_R)
		{
			soa.velocities_bb[boids[k]] = glm::vec3(0, 0, 1);

			/*
			(*boidParams.separation_component) += norm / len;
			(*boidParams.velocity_sum) += vel[j];
			(*boidParams.position_sum) += pos[j];
			(*boidParams.neighbors)++;
			*/
		}
	}
}

__device__ glm::vec3 apply_boid_rules(cudaArrays soa, const Shoal::paramsStruct& params, int i, double d)
{
	glm::vec3* pos = soa.positions;
	glm::vec3* pos_bb = soa.positions_bb;
	glm::vec3* vel = soa.velocities;
	int* grid_starts = soa.grid_starts;
	int* grid_cells = soa.grid_cells;
	int* boids = soa.grid_boids;

	glm::vec3 separation_component(0);
	glm::vec3 velocity_sum(0);
	glm::vec3 position_sum(0);
	int neighbors = 0;
	float radius_sq = MIN_R * MIN_R;

	boidParamsStruct boidParams;

	boidParams.separation_component = &separation_component;
	boidParams.velocity_sum = &velocity_sum;
	boidParams.position_sum = &position_sum;
	boidParams.neighbors = &neighbors;

#ifdef NAIVE
	for (int j = 0; j < Application::N; ++j)
	{
		glm::vec3 diff = pos[i] - pos[j];
		float len = glm::length(diff);
		glm::vec3 norm = glm::normalize(diff);

		if (i != j && len < MIN_R)
		{
			separation_component += norm / len;
			velocity_sum += vel[j];
			position_sum += pos[j];

			neighbors++;
		}
	}
#else
	int density = (int)glm::ceil(WORLD_WIDTH / GRID_R);
	int cell = calculate_grid_index(pos[i]);
	int gridX = cell % density;
	int gridY = (cell / density) % density;
	int gridZ = cell / (density * density);

	// todo: calculate this beforehand? sort a duplicate buffer?
	int start_offset, x_offset, y_offset, z_offset;

	if (pos[i].x >= LEFT_WALL + (gridX + 0.5) * GRID_R)
	{
		start_offset = 0;
		x_offset = 1;
	}
	else
	{
		start_offset = -1;
		x_offset = -1;
	}

	if (pos[i].y >= DOWN_WALL + (gridY + 0.5) * GRID_R)
		y_offset = density;
	else
		y_offset = -density;

	iterate_through_cell(soa, cell, i, boidParams);

	if ((cell + x_offset) / density == cell / density)
		iterate_through_cell(soa, cell + x_offset, i, boidParams);

	if (cell + y_offset >= 0 && cell + y_offset < density * density)
		iterate_through_cell(soa, cell + y_offset, i, boidParams);

	if (cell + y_offset + x_offset >= 0
		&& cell + y_offset + x_offset < density * density
		&& (cell + y_offset + x_offset) / density == (cell + y_offset) / density)
		iterate_through_cell(soa, cell + y_offset + x_offset, i, boidParams);
#endif

	glm::vec3 alignment_component;
	glm::vec3 cohesion_component;

	if (neighbors == 0)
		return glm::vec3(0, 0, 0);

	velocity_sum /= neighbors;
	position_sum /= neighbors;
	alignment_component = velocity_sum - vel[i];
	cohesion_component = position_sum - pos[i];

	return (float)d *
		(params.s * separation_component
			+ params.a * alignment_component
			+ params.c * cohesion_component);
}

__device__ glm::vec3 speed_limit(glm::vec3 vel, const Shoal::paramsStruct& params)
{
	if (glm::length(vel) < params.min_speed)
		return params.min_speed * glm::normalize(vel);
	if (glm::length(vel) > params.max_speed)
		return params.max_speed * glm::normalize(vel);

	return vel;
}

__device__ glm::vec3 turn_from_wall(glm::vec3 pos, glm::vec3 vel, const Shoal::paramsStruct& params)
{
	float dx_right = 1 - pos.x;
	float dx_left = pos.x + 1;
	float dy_up = 1 - pos.y;
	float dy_down = pos.y + 1;
	float dz_front = 1 - pos.z;
	float dz_back = pos.z + 1;

	float len = glm::length(vel);

	glm::vec3 vel_change = glm::vec3(0, 0, 0);

	if (dx_right < params.margin)
		vel_change.x -= params.turn * len / (dx_right * dx_right);
	if (dx_left < params.margin)
		vel_change.x += params.turn * len / (dx_left * dx_left);
	if (dy_up < params.margin)
		vel_change.y -= params.turn * len / (dy_up * dy_up);
	if (dy_down < params.margin)
		vel_change.y += params.turn * len / (dy_down * dy_down);
	if (dz_front < params.margin)
		vel_change.z -= params.turn * len / (dz_front * dz_front);
	if (dz_back < params.margin)
		vel_change.z += params.turn * len / (dz_back * dz_back);


	return vel + vel_change;
}

__device__ glm::vec3 teleport_through_wall(glm::vec3 pos)
{
	glm::vec3 ret = pos;

	if (pos.x > 1)
		ret.x = -1;
	if (pos.y > 1)
		ret.y = -1;
	if (pos.z > 1)
		ret.z = -1;

	if (pos.x < -1)
		ret.x = 1;
	if (pos.y < -1)
		ret.y = 1;
	if (pos.z < -1)
		ret.z = 1;

	return ret;
}

__global__ void calculateGridKernel(struct cudaArrays soa)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i >= Application::N)
		return;

	//printf("calculate grid\n");

	size_t density = (int)glm::ceil(WORLD_WIDTH / GRID_R);
	size_t grid_size = density * density * density * sizeof(int);

	//printf("grid size %d\n", grid_size);
	//printf("zjeb %d\n", i);

	//int d = soa.grid_cells[i];

	soa.grid_cells[i] = calculate_grid_index(soa.positions[i]);
	soa.grid_boids[i] = i;
	
	// these are getting sorted later
}

__global__ void calculateGridStartsKernel(struct cudaArrays soa)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i >= Application::N)
		return;

	//printf("calculate grid starts\n");

	size_t density = (int)glm::ceil(WORLD_WIDTH / GRID_R);
	size_t grid_size = density * density * density * sizeof(int);

	//printf("grid_size: %d\n", grid_size);

	int cell = soa.grid_cells[i];
	int prev_cell = i != 0 ? soa.grid_cells[i - 1] : -1;

	//printf("cell: %d\n", cell);

	/*
	if (i == 0 || cell != prev_cell)
	{
		// todo: there's a bottleneck
		soa.grid_starts[cell] = i;
		if (i != 0)
			soa.grid_ends[prev_cell] = i;
	}
	*/
	//printf("pa pa: %d\n", cell);
}

__global__ void calculateBoidsKernel(struct cudaArrays soa,  Shoal::paramsStruct params, double d, glm::mat4* models)
{
	printf("calcualte boids???\n");
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i >= Application::N)
		return;

	//printf("calcualte boids\n");
	//printf("%f, %f, %f\n", soa.positions[i].x, soa.positions[i].y, soa.positions[i].z);
	//printf("%f, %f, %f\n", soa.positions_bb[i].x, soa.positions_bb[i].y, soa.positions_bb[i].z);

#ifdef SNAP_TO_GRID
	glm::vec3* pos = soa.positions;
	glm::vec3* vel = soa.velocities;
	int* grid_starts = soa.grid_starts;
	int* grid_cells = soa.grid_cells;
	int* boids = soa.grid_boids;

	int cell = calculate_grid_index(pos[i]);
	int density = (int)glm::ceil(WORLD_WIDTH / GRID_R);
	
	//int jebanegowno = (int)glm::floor((pos[i].z - LEFT_WALL + 1e-3) / GRID_R);

	/*
	int gridY = (int)glm::floor((pos[i].y - DOWN_WALL + 1e-3) / GRID_R);
	*/
	int gridX = (int)glm::floor((pos[i].x - LEFT_WALL + 1e-3) / GRID_R);
	int gridZ = (int)glm::floor((pos[i].z - BACK_WALL + 1e-3) / GRID_R);
	int jebanegowno2 = (int)glm::floor((pos[i].y - DOWN_WALL + 1e-3) / GRID_R);

	//soa.positions_bb[i] = soa.positions[i];
	//soa.velocities_bb[i] = soa.velocities[i];

	//soa.positions_bb[i] = glm::vec3(LEFT_WALL + GRID_R * gridX, DOWN_WALL + GRID_R * gridY, 0);//BACK_WALL + GRID_R * gridZ);
	//else
		//soa.positions_bb[i] = glm::vec3(x, y, z);//0.7f * glm::vec3(glm::sin(d), glm::cos(d));

	//apply_boid_rules(soa, params, i, 1);
	
#else
	glm::vec3 new_vel;

	new_vel = soa.velocities[i]
		+ apply_boid_rules(soa, params, i, 1);
	new_vel = speed_limit(new_vel, params);
	new_vel = turn_from_wall(soa.positions[i], new_vel, params);

	soa.velocities_bb[i] = new_vel;
	glm::vec3 new_pos = soa.positions[i] + (float)d * new_vel;
	new_pos = teleport_through_wall(new_pos);

	soa.positions_bb[i] = new_pos;

	if (new_pos.x < -1 || new_pos.x > 1 || new_pos.y < -1 || new_pos.y > 1)
		printf("%f, %f\n", new_pos.x, new_pos.y);
#endif
	__syncthreads();

	//soa.positions[i] = soa.positions_bb[i];
	//soa.velocities[i] = soa.velocities_bb[i];

	glm::vec3 v = glm::normalize(soa.velocities[i]);

	float c1 = sqrt(v.x * v.x + v.y * v.y);
	float s1 = v.z;

	float c2 = c1 ? v.x / c1 : 1.0;
	float s2 = c1 ? v.y / c1 : 0.0;

	models[i] = glm::mat4(
		glm::vec4(v, 0),
		glm::vec4(-s2, c2, 0, 0),
		glm::vec4(-s1 * c2, -s1 * s2, c1, 0),
		glm::vec4(soa.positions[i], 1)
	);
	
	/*
	models[i] = glm::mat4(
		glm::vec4(1, 0, 0, 0),
		glm::vec4(0, 1, 0, 0),
		glm::vec4(0, 0, 1, 0),
		glm::vec4(soa.positions[i], 1)
	);
	*/
}
