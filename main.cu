﻿#include "framework.h"
#include <ctime>
#include <cstdlib>

int main()
{
	srand(time(NULL));

	initWindow();
	cpu_shoal* shoal = new cpu_shoal();

	create_buffer_objects(shoal);

	Shader shader("./vertex.glsl", "./fragment.glsl");
	shader.use();
	shader.setFloat3("boidColor", 0.9f, 0.5f, 0.0f);

	while (!glfwWindowShouldClose(window))
	{
		processInput(window);

		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		glBindVertexArray(VAO);

		glBindBuffer(GL_ARRAY_BUFFER, modelVBO);

#ifndef CPU
		size_t mat_size = N * sizeof(glm::mat3);
		size_t vec_size = N * sizeof(glm::vec2);

		glm::mat3* host_model = (glm::mat3*)malloc(mat_size);

		glm::mat3* model;
		hipMalloc(&model, mat_size);
		glm::vec2* positions;
		hipMalloc(&positions, vec_size);
		glm::vec2* velocities;
		hipMalloc(&velocities, vec_size);

		hipMemcpy(positions, shoal->positions, vec_size, hipMemcpyHostToDevice);
		hipMemcpy(velocities, shoal->velocities, vec_size, hipMemcpyHostToDevice);

		kernel_tmp << <1, N >> > (model, positions, velocities);

		hipMemcpy(host_model, model, mat_size, hipMemcpyDeviceToHost);
		glBufferData(GL_ARRAY_BUFFER, mat_size, host_model, GL_DYNAMIC_DRAW);

		free(host_model);
		hipFree(model);
		hipFree(positions);
		hipFree(velocities);
#else
		shoal->update_boids();
		glBufferData(GL_ARRAY_BUFFER, sizeof(shoal->model), &(shoal->model)[0], GL_DYNAMIC_DRAW);
#endif

		glDrawArraysInstanced(GL_TRIANGLES, 0, 6, N);
		showError();

		glfwSwapBuffers(window);
		glfwPollEvents();

	}

	glfwTerminate();
	
	//free(host_model);
	//hipFree(model);
	


	delete shoal;

	return 0;
}

