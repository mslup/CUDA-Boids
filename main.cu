#include "hip/hip_runtime.h"
﻿#include "framework.h"
#include <ctime>
#include <cstdlib>

void checkCudaError() 
{
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error: %s\n", hipGetErrorString(cudaStatus));
	}
}

void gpu(cpu_shoal*, double);

glm::mat3* model;
glm::vec2* positions;
glm::vec2* velocities;
glm::vec2* positions_bb;
glm::vec2* velocities_bb;
int* grid_cells;
int* grid_boids;
int* grid_starts;
int* grid_cellsizes;

int main()
{

#ifndef CPU
	hipSetDevice(0);
#endif

	//srand(time(NULL));
	initWindow();

	cpu_shoal* shoal = new cpu_shoal();

	create_buffer_objects(shoal);

	Shader shader("./vertex.glsl", "./fragment.glsl");
	shader.use();

#ifdef CPU
	shader.setFloat3("boidColor", 0.2f, 0.7f, 0.4f);
#else
	shader.setFloat3("boidColor", 0.9f, 0.5f, 0.0f);

	size_t mat_size = N * sizeof(glm::mat3);
	size_t vec_size = N * sizeof(glm::vec2);
	size_t int_size = N * sizeof(int);

	hipMalloc(&model, mat_size);
	hipMalloc(&positions, vec_size);
	hipMalloc(&velocities, vec_size);
	hipMalloc(&positions_bb, vec_size);
	hipMalloc(&velocities_bb, vec_size);
	hipMalloc(&grid_cells, int_size);
	hipMalloc(&grid_boids, int_size);
	hipMalloc(&grid_starts, int_size);
	hipMalloc(&grid_cellsizes, int_size);
#endif

	double previousTime = glfwGetTime();
	while (!glfwWindowShouldClose(window))
	{
		double currentTime = glfwGetTime();
		double deltaTime = currentTime - previousTime;
		previousTime = currentTime;

		processInput(window);

		ImGui_ImplOpenGL3_NewFrame();
		ImGui_ImplGlfw_NewFrame();
		ImGui::NewFrame();

		ImGui::SliderFloat("cohesion", &shoal->c, 0.0f, 0.5f);
		ImGui::SliderFloat("separation", &shoal->s, 0.0f, 0.5f);
		ImGui::SliderFloat("alignment", &shoal->a, 0.0f, 0.5f);
		ImGui::SliderFloat("max_speed", &shoal->max_speed, 0.5f, 1.0f);
		ImGui::SliderFloat("min_speed", &shoal->min_speed, 0.0f, 0.5f);
		ImGui::SliderFloat("visbility_radius", &shoal->visibility_radius, 0.0f, 0.5f);



		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		glBindVertexArray(VAO);
		glBindBuffer(GL_ARRAY_BUFFER, modelVBO);

#ifdef CPU
		shoal->update_boids(deltaTime);
		glBufferData(GL_ARRAY_BUFFER, sizeof(shoal->model), &(shoal->model)[0], GL_DYNAMIC_DRAW);
#else
		gpu(shoal, deltaTime);
#endif

		glDrawArraysInstanced(GL_TRIANGLES, 0, 6, N);
		showError();

		ImGui::Render();
		ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

		glfwSwapBuffers(window);
		glfwPollEvents();

		//for (;;);
	}

	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();
	glfwTerminate();

	delete shoal;
	hipFree(model);
	hipFree(positions);
	hipFree(positions_bb);
	hipFree(velocities);
	hipFree(velocities_bb);
	hipFree(grid_cells);
	hipFree(grid_boids);
	hipFree(grid_starts);
	hipFree(grid_cellsizes);

	// tu powinno byc jakies zwalnianie cudy

	return 0;
}

struct compare_by_x {
	__host__ __device__
		bool operator()(const glm::vec2& a, const glm::vec2& b) const {
		return a.x < b.x;
	}
};

void gpu(cpu_shoal* shoal, double deltaTime)
{
	// INITIALIZE ---------------------------------------------------------
	size_t mat_size = N * sizeof(glm::mat3);
	size_t vec_size = N * sizeof(glm::vec2);
	size_t int_size = N * sizeof(int);

	glm::mat3* host_model = (glm::mat3*)malloc(mat_size);

	hipMemcpy(positions, shoal->positions, vec_size, hipMemcpyHostToDevice);
	hipMemcpy(velocities, shoal->velocities, vec_size, hipMemcpyHostToDevice);
	hipMemcpy(positions_bb, positions, vec_size, hipMemcpyDeviceToDevice);
	hipMemcpy(velocities_bb, velocities, vec_size, hipMemcpyDeviceToDevice);

	const int max_threads = 1024;
	int blocks_per_grid = (N + max_threads - 1) / max_threads;


	// GRID ----------------------------------------------------------------
	calculateGridKernel << <blocks_per_grid, max_threads >> > (
		grid_cells, grid_boids, positions);

	/*
	thrust::sort_by_key(thrust::device, grid_cells, grid_cells + N, grid_boids);

	int* grid_cpu = (int*)malloc(int_size);
	hipMemcpy(grid_cpu, grid_cells, int_size, hipMemcpyDeviceToHost);
	int* grid_boids_cpu = (int*)malloc(int_size);
	hipMemcpy(grid_boids_cpu, grid_boids, int_size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i)
		std::cout << i << " " << grid_cpu[i] << ", " << grid_boids_cpu[i] << std::endl;

	//  do this on gpu
	size_t density = (int)glm::ceil(WORLD_WIDTH / GRID_R);
	size_t size = density * density;
	size_t num_bytes = size * sizeof(int);

	int* starts = (int*)malloc(num_bytes);
	int* sizes = (int*)malloc(num_bytes);
	std::memset(starts, -1, num_bytes);
	std::memset(sizes, -1, num_bytes);

	int start_index = 0;
	int current_cell = 0;
	int len = 1;
	int i = 0;

	while (i < size) 
	{
		int current_cell = grid_cpu[i];
		int start_index = i;
		int len = 1;  

		while (i + 1 < size && grid_cpu[i] == grid_cpu[i + 1]) {
			len++;
			i++;
		}

		sizes[current_cell] = len;
		starts[current_cell] = start_index;

		std::cout << i << " " << starts[i] << std::endl;
		i++;
	}
	
	hipMemcpy(grid_starts, starts, num_bytes, hipMemcpyHostToDevice);
	hipMemcpy(grid_cellsizes, sizes, num_bytes, hipMemcpyHostToDevice);

	free(grid_boids_cpu);
	free(grid_cpu);
	free(starts);
	free(sizes);

	*/

	// POSITIONS & VELOCITIES -----------------------------------------------
	calculateBoidsKernel << <blocks_per_grid, max_threads >> > (
		positions, velocities,
		positions_bb, velocities_bb,
		grid_cells, grid_boids,
		deltaTime);

	hipMemcpy(positions, positions_bb, vec_size, hipMemcpyDeviceToDevice);
	hipMemcpy(velocities, velocities_bb, vec_size, hipMemcpyDeviceToDevice);

	// MODEL MATRICES -------------------------------------------------------
	calculateModelKernel << <blocks_per_grid, max_threads >> > (model, positions, velocities);

	hipMemcpy(shoal->positions, positions_bb, vec_size, hipMemcpyDeviceToHost);
	hipMemcpy(shoal->velocities, velocities_bb, vec_size, hipMemcpyDeviceToHost);

	hipMemcpy(host_model, model, mat_size, hipMemcpyDeviceToHost);
	glBufferData(GL_ARRAY_BUFFER, mat_size, host_model, GL_DYNAMIC_DRAW);

	free(host_model);
}
