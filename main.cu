﻿#include "framework.h"
#include <ctime>
#include <cstdlib>

int main()
{

#ifndef CPU
	hipSetDevice(0);
#endif

	srand(time(NULL));

	initWindow();
	cpu_shoal* shoal = new cpu_shoal();

	create_buffer_objects(shoal);

	Shader shader("./vertex.glsl", "./fragment.glsl");
	shader.use();
	shader.setFloat3("boidColor", 0.9f, 0.5f, 0.0f);

	double previousTime = glfwGetTime();
	while (!glfwWindowShouldClose(window))
	{
		double currentTime = glfwGetTime(); //TODO: platform
		double deltaTime = currentTime - previousTime;
		previousTime = currentTime;

		processInput(window);

		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		glBindVertexArray(VAO);

		glBindBuffer(GL_ARRAY_BUFFER, modelVBO);

#ifndef CPU
		size_t mat_size = N * sizeof(glm::mat3);
		size_t vec_size = N * sizeof(glm::vec2);

		glm::mat3* host_model = (glm::mat3*)malloc(mat_size);

		glm::mat3* model;
		hipMalloc(&model, mat_size);
		glm::vec2* positions;
		hipMalloc(&positions, vec_size);
		glm::vec2* velocities;
		hipMalloc(&velocities, vec_size);
		glm::vec2* positions_bb;
		hipMalloc(&positions_bb, vec_size);
		glm::vec2* velocities_bb;
		hipMalloc(&velocities_bb, vec_size);

		hipMemcpy(positions, shoal->positions, vec_size, hipMemcpyHostToDevice);
		hipMemcpy(velocities, shoal->velocities, vec_size, hipMemcpyHostToDevice);

		calculateBoidsKernel << <1, N >> > (positions, velocities, 
			positions_bb, velocities_bb, deltaTime);

		hipMemcpy(positions, positions_bb, vec_size, hipMemcpyDeviceToDevice);
		hipMemcpy(velocities, velocities_bb, vec_size, hipMemcpyDeviceToDevice);
		
		calculateModelKernel << <1, N >> > (model, positions, velocities);

		hipMemcpy(shoal->positions, positions_bb, vec_size, hipMemcpyDeviceToHost);
		hipMemcpy(shoal->velocities, velocities_bb, vec_size, hipMemcpyDeviceToHost);

		//std::cout << shoal->positions[0].x << std::endl;

		hipMemcpy(host_model, model, mat_size, hipMemcpyDeviceToHost);
		glBufferData(GL_ARRAY_BUFFER, mat_size, host_model, GL_DYNAMIC_DRAW);

		free(host_model);
		hipFree(model);
		hipFree(positions);
		hipFree(positions_bb);
		hipFree(velocities);
		hipFree(velocities_bb);
#else
		shoal->update_boids();
		glBufferData(GL_ARRAY_BUFFER, sizeof(shoal->model), &(shoal->model)[0], GL_DYNAMIC_DRAW);
#endif

		glDrawArraysInstanced(GL_TRIANGLES, 0, 6, N);
		showError();

		glfwSwapBuffers(window);
		glfwPollEvents();

	}

	glfwTerminate();
	
	delete shoal;

	return 0;
}

